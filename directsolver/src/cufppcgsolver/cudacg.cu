#include "hip/hip_runtime.h"
#include "cudacg.h"
// CUDA Runtime
#include <hip/hip_runtime.h>

// Using updated (v2) interfaces for CUBLAS and CUSPARSE
#include <hipsparse.h>
#include <hipblas.h>

// Utilities and system includes
//#include "cudasamples/helper_functions.h"
//#include "cudasamples/hip/hip_runtime_api.h"
#include <iostream>
#include <memory>
#include <vector>
#include <numeric>
#include <algorithm>
#include <chrono>
// Market Matrix I/O
extern "C" {
#include "mm/mmio.h"
}

std::tuple<long, long, int> runCusparseCublasCG(std::vector<int> &I, std::vector<int> &J, std::vector<double> &val, std::vector<double> &rhs, std::vector<double> &x, int M, int N, int nz, double tol, int max_iter) {
	int *d_col, *d_row;
	double r0, r1, alpha, beta;
	double *d_val, *d_x;
	double *d_zm1, *d_zm2, *d_rm2;
	double *d_r, *d_p, *d_omega, *d_y;
	double *d_valsILU0;
	void *buffer = NULL;
	double dot, numerator, denominator, nalpha;
	const double floatone = 1.0;
	const double floatzero = 0.0;

	int nErrors = 0;

	auto t1 = std::chrono::high_resolution_clock::now();
	/* Create CUBLAS context */
	hipblasHandle_t cublasHandle = 0;
	hipblasStatus_t hipblasStatus_t;
	hipblasStatus_t = hipblasCreate(&cublasHandle);

	/* Create CUSPARSE context */
	hipsparseHandle_t cusparseHandle = 0;
	hipsparseStatus_t cusparseStatus;
	cusparseStatus = hipsparseCreate(&cusparseHandle);

	/* Description of the A matrix*/
	hipsparseMatDescr_t descr = 0;
	cusparseStatus = hipsparseCreateMatDescr(&descr);
	hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

	/* Allocate required memory */
	hipMalloc((void **)&d_col, nz * sizeof(int));
	hipMalloc((void **)&d_row, (N + 1) * sizeof(int));
	hipMalloc((void **)&d_val, nz * sizeof(double));
	hipMalloc((void **)&d_x, N * sizeof(double));
	hipMalloc((void **)&d_y, N * sizeof(double));
	hipMalloc((void **)&d_r, N * sizeof(double));
	hipMalloc((void **)&d_p, N * sizeof(double));
	hipMalloc((void **)&d_omega, N * sizeof(double));
	hipMalloc((void **)&d_valsILU0, nz * sizeof(double));
	hipMalloc((void **)&d_zm1, (N) * sizeof(double));
	hipMalloc((void **)&d_zm2, (N) * sizeof(double));
	hipMalloc((void **)&d_rm2, (N) * sizeof(double));

	/* Wrap raw data into cuSPARSE generic API objects */
	hipsparseSpMatDescr_t matA = NULL;
	cusparseStatus = hipsparseCreateCsr(&matA, N, N, nz, d_row, d_col, d_val,
									HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
									HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);
	hipsparseDnVecDescr_t vecp = NULL;
	cusparseStatus = hipsparseCreateDnVec(&vecp, N, d_p, HIP_R_64F);
	hipsparseDnVecDescr_t vecomega = NULL;
	cusparseStatus = hipsparseCreateDnVec(&vecomega, N, d_omega, HIP_R_64F);

	/* Initialize problem data */
	hipMemcpy(d_col, J.data(), nz * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_row, I.data(), (N + 1) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_val, val.data(), nz * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_x, x.data(), N * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_r, rhs.data(), N * sizeof(double), hipMemcpyHostToDevice);

	/* Create ILU(0) info object */
	csrilu02Info_t infoILU = NULL;
	cusparseStatus = hipsparseCreateCsrilu02Info(&infoILU);

	/* Create L factor descriptor and triangular solve info */
	hipsparseMatDescr_t descrL = NULL;
	cusparseStatus = hipsparseCreateMatDescr(&descrL);
	cusparseStatus = hipsparseSetMatType(descrL, HIPSPARSE_MATRIX_TYPE_GENERAL);
	cusparseStatus = hipsparseSetMatIndexBase(descrL, HIPSPARSE_INDEX_BASE_ZERO);
	cusparseStatus = hipsparseSetMatFillMode(descrL, HIPSPARSE_FILL_MODE_LOWER);
	cusparseStatus = hipsparseSetMatDiagType(descrL, HIPSPARSE_DIAG_TYPE_UNIT);
	csrsv2Info_t infoL = NULL;
	cusparseStatus = hipsparseCreateCsrsv2Info(&infoL);
  
	/* Create U factor descriptor and triangular solve info */
	hipsparseMatDescr_t descrU = NULL;
	cusparseStatus = hipsparseCreateMatDescr(&descrU);
	cusparseStatus = hipsparseSetMatType(descrU, HIPSPARSE_MATRIX_TYPE_GENERAL);
	cusparseStatus = hipsparseSetMatIndexBase(descrU, HIPSPARSE_INDEX_BASE_ZERO);
	cusparseStatus = hipsparseSetMatFillMode(descrU, HIPSPARSE_FILL_MODE_UPPER);
	cusparseStatus = hipsparseSetMatDiagType(descrU, HIPSPARSE_DIAG_TYPE_NON_UNIT);
	csrsv2Info_t infoU = NULL;
	cusparseStatus = hipsparseCreateCsrsv2Info(&infoU);
  
	/* Allocate workspace for cuSPARSE */
	size_t bufferSize = 0;
	size_t tmp = 0;
	int stmp = 0;
	cusparseStatus = hipsparseSpMV_bufferSize(
		cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &floatone, matA, vecp,
		&floatzero, vecomega, HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT, &tmp);
	if (tmp > bufferSize) {
	  bufferSize = stmp;
	}
	cusparseStatus = hipsparseDcsrilu02_bufferSize(
		cusparseHandle, N, nz, descr, d_val, d_row, d_col, infoILU, &stmp);
	if (stmp > bufferSize) {
	  bufferSize = stmp;
	}
	cusparseStatus = hipsparseDcsrsv2_bufferSize(
		cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, nz, descrL, d_val,
		d_row, d_col, infoL, &stmp);
	if (stmp > bufferSize) {
	  bufferSize = stmp;
	}
	cusparseStatus = hipsparseDcsrsv2_bufferSize(
		cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, nz, descrU, d_val,
		d_row, d_col, infoU, &stmp);
	if (stmp > bufferSize) {
	  bufferSize = stmp;
	}
	hipMalloc(&buffer, bufferSize);

	/* Preconditioned Conjugate Gradient using ILU.
	--------------------------------------------
	Follows the description by Golub & Van Loan, "Matrix Computations 3rd ed.", Algorithm 10.3.1  */

	/* Perform analysis for ILU(0) */
	cusparseStatus = hipsparseDcsrilu02_analysis(
		cusparseHandle, N, nz, descr, d_val, d_row, d_col, infoILU,
		HIPSPARSE_SOLVE_POLICY_USE_LEVEL, buffer);
  
	/* Copy A data to ILU(0) vals as input*/
	hipMemcpy(d_valsILU0, d_val, nz * sizeof(double),
							   hipMemcpyDeviceToDevice);
  
	/* generate the ILU(0) factors */
	cusparseStatus = hipsparseDcsrilu02(cusparseHandle, N, nz, descr, d_valsILU0,
									  d_row, d_col, infoILU,
									  HIPSPARSE_SOLVE_POLICY_USE_LEVEL, buffer);

	auto t2 = std::chrono::high_resolution_clock::now();
	std::chrono::duration<double> time_analyser = t2 - t1;
	std::cout << "Cublas analyser on A used " << std::chrono::duration_cast<std::chrono::microseconds>(time_analyser).count() << " us." << std::endl;
  
	/* perform triangular solve analysis */
	cusparseStatus = hipsparseDcsrsv2_analysis(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
								N, nz, descrL, d_valsILU0, d_row, d_col, infoL,
								HIPSPARSE_SOLVE_POLICY_USE_LEVEL, buffer);

	cusparseStatus = hipsparseDcsrsv2_analysis(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
								N, nz, descrU, d_valsILU0, d_row, d_col, infoU,
								HIPSPARSE_SOLVE_POLICY_USE_LEVEL, buffer);

	/* reset the initial guess of the solution to zero */
	for (int i = 0; i < N; i++) x[i] = 0.0;

	hipMemcpy(d_r, rhs.data(), N * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_x, x.data(), N * sizeof(double), hipMemcpyHostToDevice);

	t1 = std::chrono::high_resolution_clock::now();
	int k = 0;
	hipblasDdot(cublasHandle, N, d_r, 1, d_r, 1, &r1);

#ifdef CGTIMING
	double totalItTime, totalTriangularTime, totalSpmvTime;
	totalItTime = totalTriangularTime = totalSpmvTime = 0;
	hipEvent_t startTotal, stopTotal, startTri, stopTri, startSpmv, stopSpmv;
	hipEventCreate(&startTotal); hipEventCreate(&stopTotal);
	hipEventCreate(&startTri); hipEventCreate(&stopTri);
	hipEventCreate(&startSpmv); hipEventCreate(&stopSpmv);
#endif // CGTIMING

	double tolsqr = tol > 0 ? tol*tol : -1;
	while (r1 > tolsqr && k <= max_iter)
	{
#ifdef CGTIMING
		hipEventRecord(startTotal);
		hipEventRecord(startTri);
#endif // CGTIMING
		// preconditioner application: d_zm1 = U^-1 L^-1 d_r
		cusparseStatus = hipsparseDcsrsv2_solve(
			cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, nz, &floatone,
			descrL, d_valsILU0, d_row, d_col, infoL, d_r, d_y,
			HIPSPARSE_SOLVE_POLICY_USE_LEVEL, buffer);
		cusparseStatus = hipsparseDcsrsv2_solve(
			cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, nz, &floatone,
			descrU, d_valsILU0, d_row, d_col, infoU, d_y, d_zm1,
			HIPSPARSE_SOLVE_POLICY_USE_LEVEL, buffer);
#ifdef CGTIMING
		hipEventRecord(stopTri);
#endif // CGTIMING

		k++;

		if (k == 1)
		{
			hipblasDcopy(cublasHandle, N, d_zm1, 1, d_p, 1);
		}
		else
		{
			hipblasDdot(cublasHandle, N, d_r, 1, d_zm1, 1, &numerator);
			hipblasDdot(cublasHandle, N, d_rm2, 1, d_zm2, 1, &denominator);
			beta = numerator / denominator;
			hipblasDscal(cublasHandle, N, &beta, d_p, 1);
			hipblasDaxpy(cublasHandle, N, &floatone, d_zm1, 1, d_p, 1);
		}
#ifdef CGTIMING
		hipEventRecord(startSpmv);
#endif // CGTIMING
		hipsparseSpMV(
			cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &floatone, matA, vecp,
			&floatzero, vecomega, HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT, buffer);
#ifdef CGTIMING
		hipEventRecord(stopSpmv);
#endif // CGTIMING
		hipblasDdot(cublasHandle, N, d_r, 1, d_zm1, 1, &numerator);
		hipblasDdot(cublasHandle, N, d_p, 1, d_omega, 1, &denominator);
		alpha = numerator / denominator;
		hipblasDaxpy(cublasHandle, N, &alpha, d_p, 1, d_x, 1);
		hipblasDcopy(cublasHandle, N, d_r, 1, d_rm2, 1);
		hipblasDcopy(cublasHandle, N, d_zm1, 1, d_zm2, 1);
		nalpha = -alpha;
		hipblasDaxpy(cublasHandle, N, &nalpha, d_omega, 1, d_r, 1);
		hipblasDdot(cublasHandle, N, d_r, 1, d_r, 1, &r1);
#ifdef CGTIMING
		hipEventRecord(stopTotal);
#endif // CGTIMING
#ifdef CGTIMING
		hipEventSynchronize(stopTotal); hipEventSynchronize(stopTri); hipEventSynchronize(stopSpmv);
		float msTotal, msTri, msSpmv;  msTotal = msTri = msSpmv = 0;
		hipEventElapsedTime(&msTotal, startTotal, stopTotal); hipEventElapsedTime(&msTri, startTri, stopTri); hipEventElapsedTime(&msSpmv, startSpmv, stopSpmv);
		totalItTime += (double)(1e3 * msTotal);
		totalTriangularTime += (double)(1e3 * msTri);
		totalSpmvTime += (double)(1e3 *  msSpmv);
#endif // CGTIMING
	}
	hipMemcpy(x.data(), d_x, N * sizeof(double), hipMemcpyDeviceToHost); 
	t2 = std::chrono::high_resolution_clock::now();

	///************************/
	///* now write out result */
	///************************/
	std::chrono::duration<double> time_executor = t2 - t1;
	std::cout << "Cublas executor on A used " << std::chrono::duration_cast<std::chrono::microseconds>(time_executor).count() << " us. Final residual is " << sqrt(r1) << " after " << k << " iterations." << std::endl;

#ifdef CGTIMING
	totalItTime /= (double)k; totalTriangularTime /= (double)k; totalSpmvTime /= (double)k;
	std::cout << "Average cublas/cusparse iteration time breakdown: " << totalTriangularTime << " (triangular solver) " << totalSpmvTime << " (spmv) " << totalItTime - totalTriangularTime - totalSpmvTime << " (remaining) " << totalItTime << " (total)." << std::endl;
#endif // CGTIMING

	/* Destroy descriptors */
	hipsparseDestroyCsrsv2Info(infoU);
	hipsparseDestroyCsrsv2Info(infoL);
	hipsparseDestroyCsrilu02Info(infoILU);
	hipsparseDestroyMatDescr(descrL);
	hipsparseDestroyMatDescr(descrU);
	hipsparseDestroyMatDescr(descr);
	hipsparseDestroySpMat(matA);
	hipsparseDestroyDnVec(vecp);
	hipsparseDestroyDnVec(vecomega);

	/* Destroy contexts */
	hipsparseDestroy(cusparseHandle);
	hipblasDestroy(cublasHandle);

	/* Free device memory */
	hipFree(d_col);
	hipFree(d_row);
	hipFree(d_val);
	hipFree(d_x);
	hipFree(d_y);
	hipFree(d_r);
	hipFree(d_p);
	hipFree(d_omega);
	hipFree(d_valsILU0);
	hipFree(d_zm1);
	hipFree(d_zm2);
	hipFree(d_rm2);

	return std::make_tuple(std::chrono::duration_cast<std::chrono::microseconds>(time_analyser).count(), std::chrono::duration_cast<std::chrono::microseconds>(time_executor).count(), k);
}