#include "hip/hip_runtime.h"
#include "cudacg.h"
// CUDA Runtime
#include <hip/hip_runtime.h>

// Using updated (v2) interfaces for CUBLAS and CUSPARSE
#include <hipsparse.h>
#include <hipblas.h>

// Utilities and system includes
//#include "cudasamples/helper_functions.h"
//#include "cudasamples/hip/hip_runtime_api.h"
#include <iostream>
#include <memory>
#include <vector>
#include <numeric>
#include <algorithm>
#include <chrono>
// Market Matrix I/O
extern "C" {
#include "mm/mmio.h"
}

std::tuple<long, long, int> runCusparseCublasCG(std::vector<int> &I, std::vector<int> &J, std::vector<float> &val, std::vector<float> &rhs, std::vector<float> &x, int M, int N, int nz, float tol, int max_iter) {
	int *d_col, *d_row;
	float r0, r1, alpha, beta;
	float *d_val, *d_x;
	float *d_zm1, *d_zm2, *d_rm2;
	float *d_r, *d_p, *d_omega, *d_y;
	float *d_valsILU0;
	void *buffer = NULL;
	float dot, numerator, denominator, nalpha;
	const float floatone = 1.0;
	const float floatzero = 0.0;

	int nErrors = 0;

	auto t1 = std::chrono::high_resolution_clock::now();
	/* Create CUBLAS context */
	hipblasHandle_t cublasHandle = 0;
	hipblasStatus_t hipblasStatus_t;
	hipblasStatus_t = hipblasCreate(&cublasHandle);

	/* Create CUSPARSE context */
	hipsparseHandle_t cusparseHandle = 0;
	hipsparseStatus_t cusparseStatus;
	cusparseStatus = hipsparseCreate(&cusparseHandle);

	/* Description of the A matrix*/
	hipsparseMatDescr_t descr = 0;
	cusparseStatus = hipsparseCreateMatDescr(&descr);
	hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

	/* Allocate required memory */
	hipMalloc((void **)&d_col, nz * sizeof(int));
	hipMalloc((void **)&d_row, (N + 1) * sizeof(int));
	hipMalloc((void **)&d_val, nz * sizeof(float));
	hipMalloc((void **)&d_x, N * sizeof(float));
	hipMalloc((void **)&d_y, N * sizeof(float));
	hipMalloc((void **)&d_r, N * sizeof(float));
	hipMalloc((void **)&d_p, N * sizeof(float));
	hipMalloc((void **)&d_omega, N * sizeof(float));
	hipMalloc((void **)&d_valsILU0, nz * sizeof(float));
	hipMalloc((void **)&d_zm1, (N) * sizeof(float));
	hipMalloc((void **)&d_zm2, (N) * sizeof(float));
	hipMalloc((void **)&d_rm2, (N) * sizeof(float));

	/* Wrap raw data into cuSPARSE generic API objects */
	hipsparseSpMatDescr_t matA = NULL;
	cusparseStatus = hipsparseCreateCsr(&matA, N, N, nz, d_row, d_col, d_val,
									HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
									HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);
	hipsparseDnVecDescr_t vecp = NULL;
	cusparseStatus = hipsparseCreateDnVec(&vecp, N, d_p, HIP_R_32F);
	hipsparseDnVecDescr_t vecomega = NULL;
	cusparseStatus = hipsparseCreateDnVec(&vecomega, N, d_omega, HIP_R_32F);

	/* Initialize problem data */
	hipMemcpy(d_col, J.data(), nz * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_row, I.data(), (N + 1) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_val, val.data(), nz * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_x, x.data(), N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_r, rhs.data(), N * sizeof(float), hipMemcpyHostToDevice);

	/* Create ILU(0) info object */
	csrilu02Info_t infoILU = NULL;
	cusparseStatus = hipsparseCreateCsrilu02Info(&infoILU);

	/* Create L factor descriptor and triangular solve info */
	hipsparseMatDescr_t descrL = NULL;
	cusparseStatus = hipsparseCreateMatDescr(&descrL);
	cusparseStatus = hipsparseSetMatType(descrL, HIPSPARSE_MATRIX_TYPE_GENERAL);
	cusparseStatus = hipsparseSetMatIndexBase(descrL, HIPSPARSE_INDEX_BASE_ZERO);
	cusparseStatus = hipsparseSetMatFillMode(descrL, HIPSPARSE_FILL_MODE_LOWER);
	cusparseStatus = hipsparseSetMatDiagType(descrL, HIPSPARSE_DIAG_TYPE_UNIT);
	csrsv2Info_t infoL = NULL;
	cusparseStatus = hipsparseCreateCsrsv2Info(&infoL);
  
	/* Create U factor descriptor and triangular solve info */
	hipsparseMatDescr_t descrU = NULL;
	cusparseStatus = hipsparseCreateMatDescr(&descrU);
	cusparseStatus = hipsparseSetMatType(descrU, HIPSPARSE_MATRIX_TYPE_GENERAL);
	cusparseStatus = hipsparseSetMatIndexBase(descrU, HIPSPARSE_INDEX_BASE_ZERO);
	cusparseStatus = hipsparseSetMatFillMode(descrU, HIPSPARSE_FILL_MODE_UPPER);
	cusparseStatus = hipsparseSetMatDiagType(descrU, HIPSPARSE_DIAG_TYPE_NON_UNIT);
	csrsv2Info_t infoU = NULL;
	cusparseStatus = hipsparseCreateCsrsv2Info(&infoU);
  
	/* Allocate workspace for cuSPARSE */
	size_t bufferSize = 0;
	size_t tmp = 0;
	int stmp = 0;
	cusparseStatus = hipsparseSpMV_bufferSize(
		cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &floatone, matA, vecp,
		&floatzero, vecomega, HIP_R_32F, HIPSPARSE_SPMV_ALG_DEFAULT, &tmp);
	if (tmp > bufferSize) {
	  bufferSize = stmp;
	}
	cusparseStatus = hipsparseScsrilu02_bufferSize(
		cusparseHandle, N, nz, descr, d_val, d_row, d_col, infoILU, &stmp);
	if (stmp > bufferSize) {
	  bufferSize = stmp;
	}
	cusparseStatus = hipsparseScsrsv2_bufferSize(
		cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, nz, descrL, d_val,
		d_row, d_col, infoL, &stmp);
	if (stmp > bufferSize) {
	  bufferSize = stmp;
	}
	cusparseStatus = hipsparseScsrsv2_bufferSize(
		cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, nz, descrU, d_val,
		d_row, d_col, infoU, &stmp);
	if (stmp > bufferSize) {
	  bufferSize = stmp;
	}
	hipMalloc(&buffer, bufferSize);

	/* Preconditioned Conjugate Gradient using ILU.
	--------------------------------------------
	Follows the description by Golub & Van Loan, "Matrix Computations 3rd ed.", Algorithm 10.3.1  */

	/* Perform analysis for ILU(0) */
	cusparseStatus = hipsparseScsrilu02_analysis(
		cusparseHandle, N, nz, descr, d_val, d_row, d_col, infoILU,
		HIPSPARSE_SOLVE_POLICY_USE_LEVEL, buffer);
  
	/* Copy A data to ILU(0) vals as input*/
	hipMemcpy(d_valsILU0, d_val, nz * sizeof(float),
							   hipMemcpyDeviceToDevice);
  
	/* generate the ILU(0) factors */
	cusparseStatus = hipsparseScsrilu02(cusparseHandle, N, nz, descr, d_valsILU0,
									  d_row, d_col, infoILU,
									  HIPSPARSE_SOLVE_POLICY_USE_LEVEL, buffer);

	auto t2 = std::chrono::high_resolution_clock::now();
	std::chrono::duration<double> time_analyser = t2 - t1;
	std::cout << "Cublas analyser on A used " << std::chrono::duration_cast<std::chrono::microseconds>(time_analyser).count() << " us." << std::endl;
  
	/* perform triangular solve analysis */
	cusparseStatus = hipsparseScsrsv2_analysis(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
								N, nz, descrL, d_valsILU0, d_row, d_col, infoL,
								HIPSPARSE_SOLVE_POLICY_USE_LEVEL, buffer);

	cusparseStatus = hipsparseScsrsv2_analysis(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
								N, nz, descrU, d_valsILU0, d_row, d_col, infoU,
								HIPSPARSE_SOLVE_POLICY_USE_LEVEL, buffer);

	/* reset the initial guess of the solution to zero */
	for (int i = 0; i < N; i++) x[i] = 0.0;

	hipMemcpy(d_r, rhs.data(), N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_x, x.data(), N * sizeof(float), hipMemcpyHostToDevice);

	t1 = std::chrono::high_resolution_clock::now();
	int k = 0;
	hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, &r1);

#ifdef CGTIMING
	double totalItTime, totalTriangularTime, totalSpmvTime;
	totalItTime = totalTriangularTime = totalSpmvTime = 0;
	hipEvent_t startTotal, stopTotal, startTri, stopTri, startSpmv, stopSpmv;
	hipEventCreate(&startTotal); hipEventCreate(&stopTotal);
	hipEventCreate(&startTri); hipEventCreate(&stopTri);
	hipEventCreate(&startSpmv); hipEventCreate(&stopSpmv);
#endif // CGTIMING

	float tolsqr = tol > 0 ? tol*tol : -1;
	while (r1 > tolsqr && k <= max_iter)
	{
#ifdef CGTIMING
		hipEventRecord(startTotal);
		hipEventRecord(startTri);
#endif // CGTIMING
		// preconditioner application: d_zm1 = U^-1 L^-1 d_r
		cusparseStatus = hipsparseScsrsv2_solve(
			cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, nz, &floatone,
			descrL, d_valsILU0, d_row, d_col, infoL, d_r, d_y,
			HIPSPARSE_SOLVE_POLICY_USE_LEVEL, buffer);
		cusparseStatus = hipsparseScsrsv2_solve(
			cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, nz, &floatone,
			descrU, d_valsILU0, d_row, d_col, infoU, d_y, d_zm1,
			HIPSPARSE_SOLVE_POLICY_USE_LEVEL, buffer);
#ifdef CGTIMING
		hipEventRecord(stopTri);
#endif // CGTIMING

		k++;

		if (k == 1)
		{
			hipblasScopy(cublasHandle, N, d_zm1, 1, d_p, 1);
		}
		else
		{
			hipblasSdot(cublasHandle, N, d_r, 1, d_zm1, 1, &numerator);
			hipblasSdot(cublasHandle, N, d_rm2, 1, d_zm2, 1, &denominator);
			beta = numerator / denominator;
			hipblasSscal(cublasHandle, N, &beta, d_p, 1);
			hipblasSaxpy(cublasHandle, N, &floatone, d_zm1, 1, d_p, 1);
		}
#ifdef CGTIMING
		hipEventRecord(startSpmv);
#endif // CGTIMING
		hipsparseSpMV(
			cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &floatone, matA, vecp,
			&floatzero, vecomega, HIP_R_32F, HIPSPARSE_SPMV_ALG_DEFAULT, buffer);
#ifdef CGTIMING
		hipEventRecord(stopSpmv);
#endif // CGTIMING
		hipblasSdot(cublasHandle, N, d_r, 1, d_zm1, 1, &numerator);
		hipblasSdot(cublasHandle, N, d_p, 1, d_omega, 1, &denominator);
		alpha = numerator / denominator;
		hipblasSaxpy(cublasHandle, N, &alpha, d_p, 1, d_x, 1);
		hipblasScopy(cublasHandle, N, d_r, 1, d_rm2, 1);
		hipblasScopy(cublasHandle, N, d_zm1, 1, d_zm2, 1);
		nalpha = -alpha;
		hipblasSaxpy(cublasHandle, N, &nalpha, d_omega, 1, d_r, 1);
		hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, &r1);
#ifdef CGTIMING
		hipEventRecord(stopTotal);
#endif // CGTIMING
#ifdef CGTIMING
		hipEventSynchronize(stopTotal); hipEventSynchronize(stopTri); hipEventSynchronize(stopSpmv);
		float msTotal, msTri, msSpmv;  msTotal = msTri = msSpmv = 0;
		hipEventElapsedTime(&msTotal, startTotal, stopTotal); hipEventElapsedTime(&msTri, startTri, stopTri); hipEventElapsedTime(&msSpmv, startSpmv, stopSpmv);
		totalItTime += (float)(1e3 * msTotal);
		totalTriangularTime += (float)(1e3 * msTri);
		totalSpmvTime += (float)(1e3 *  msSpmv);
#endif // CGTIMING
	}
	hipMemcpy(x.data(), d_x, N * sizeof(float), hipMemcpyDeviceToHost); 
	t2 = std::chrono::high_resolution_clock::now();

	///************************/
	///* now write out result */
	///************************/
	std::chrono::duration<double> time_executor = t2 - t1;
	std::cout << "Cublas executor on A used " << std::chrono::duration_cast<std::chrono::microseconds>(time_executor).count() << " us. Final residual is " << sqrt(r1) << " after " << k << " iterations." << std::endl;

#ifdef CGTIMING
	totalItTime /= (double)k; totalTriangularTime /= (double)k; totalSpmvTime /= (double)k;
	std::cout << "Average cublas/cusparse iteration time breakdown: " << totalTriangularTime << " (triangular solver) " << totalSpmvTime << " (spmv) " << totalItTime - totalTriangularTime - totalSpmvTime << " (remaining) " << totalItTime << " (total)." << std::endl;
#endif // CGTIMING

	/* Destroy descriptors */
	hipsparseDestroyCsrsv2Info(infoU);
	hipsparseDestroyCsrsv2Info(infoL);
	hipsparseDestroyCsrilu02Info(infoILU);
	hipsparseDestroyMatDescr(descrL);
	hipsparseDestroyMatDescr(descrU);
	hipsparseDestroyMatDescr(descr);
	hipsparseDestroySpMat(matA);
	hipsparseDestroyDnVec(vecp);
	hipsparseDestroyDnVec(vecomega);

	/* Destroy contexts */
	hipsparseDestroy(cusparseHandle);
	hipblasDestroy(cublasHandle);

	/* Free device memory */
	hipFree(d_col);
	hipFree(d_row);
	hipFree(d_val);
	hipFree(d_x);
	hipFree(d_y);
	hipFree(d_r);
	hipFree(d_p);
	hipFree(d_omega);
	hipFree(d_valsILU0);
	hipFree(d_zm1);
	hipFree(d_zm2);
	hipFree(d_rm2);

	return std::make_tuple(std::chrono::duration_cast<std::chrono::microseconds>(time_analyser).count(), std::chrono::duration_cast<std::chrono::microseconds>(time_executor).count(), k);
}