#include "hip/hip_runtime.h"
#include "cudacg.h"
// CUDA Runtime
#include <hip/hip_runtime.h>

// Using updated (v2) interfaces for CUBLAS and CUSPARSE
#include <hipsparse.h>
#include <hipblas.h>

// Utilities and system includes
//#include "cudasamples/helper_functions.h"
//#include "cudasamples/hip/hip_runtime_api.h"
#include <iostream>
#include <memory>
#include <vector>
#include <numeric>
#include <algorithm>
#include <chrono>
// Market Matrix I/O
extern "C" {
#include "mm/mmio.h"
}

std::tuple<long, long> runCusparseCublasCG(std::vector<int> &I, std::vector<int> &J, std::vector<float> &val, std::vector<float> &rhs, std::vector<float> &x, int M, int N, int nz, float tol, int max_iter) {
	int *d_col, *d_row;
	float r0, r1, alpha, beta;
	float *d_val, *d_x;
	float *d_zm1, *d_zm2, *d_rm2;
	float *d_r, *d_p, *d_omega, *d_y;
	float *d_valsILU0;
	float *valsILU0;
	float dot, numerator, denominator, nalpha;
	const float floatone = 1.0;
	const float floatzero = 0.0;

	int nErrors = 0;

	auto t1 = std::chrono::high_resolution_clock::now();
	/* Create CUBLAS context */
	hipblasHandle_t cublasHandle = 0;
	hipblasStatus_t hipblasStatus_t;
	hipblasStatus_t = hipblasCreate(&cublasHandle);

	hipblasStatus_t;

	/* Create CUSPARSE context */
	hipsparseHandle_t cusparseHandle = 0;
	hipsparseStatus_t cusparseStatus;
	cusparseStatus = hipsparseCreate(&cusparseHandle);

	cusparseStatus;

	/* Description of the A matrix*/
	hipsparseMatDescr_t descr = 0;
	cusparseStatus = hipsparseCreateMatDescr(&descr);

	cusparseStatus;

	/* Define the properties of the matrix */
	hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

	/* Allocate required memory */
	hipMalloc((void **)&d_col, nz * sizeof(int));
	hipMalloc((void **)&d_row, (N + 1) * sizeof(int));
	hipMalloc((void **)&d_val, nz * sizeof(float));
	hipMalloc((void **)&d_x, N * sizeof(float));
	hipMalloc((void **)&d_y, N * sizeof(float));
	hipMalloc((void **)&d_r, N * sizeof(float));
	hipMalloc((void **)&d_p, N * sizeof(float));
	hipMalloc((void **)&d_omega, N * sizeof(float));

	hipMemcpy(d_col, J.data(), nz * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_row, I.data(), (N + 1) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_val, val.data(), nz * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_x, x.data(), N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_r, rhs.data(), N * sizeof(float), hipMemcpyHostToDevice);

	/* Preconditioned Conjugate Gradient using ILU.
	--------------------------------------------
	Follows the description by Golub & Van Loan, "Matrix Computations 3rd ed.", Algorithm 10.3.1  */

	int nzILU0 = 2 * N - 1;
	valsILU0 = (float *)malloc(nz * sizeof(float));

	hipMalloc((void **)&d_valsILU0, nz * sizeof(float));
	hipMalloc((void **)&d_zm1, (N) * sizeof(float));
	hipMalloc((void **)&d_zm2, (N) * sizeof(float));
	hipMalloc((void **)&d_rm2, (N) * sizeof(float));

	/* create the analysis info object for the A matrix */
	cusparseSolveAnalysisInfo_t infoA = 0;
	cusparseStatus = cusparseCreateSolveAnalysisInfo(&infoA);

	/* Perform the analysis for the Non-Transpose case */
	cusparseStatus = cusparseScsrsv_analysis(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
		N, nz, descr, d_val, d_row, d_col, infoA);

	/* Copy A data to ILU0 vals as input*/
	hipMemcpy(d_valsILU0, d_val, nz * sizeof(float), hipMemcpyDeviceToDevice);

	/* generate the Incomplete LU factor H for the matrix A using cudsparseScsrilu0 */
	cusparseStatus = cusparseScsrilu0(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, descr, d_valsILU0, d_row, d_col, infoA);
	auto t2 = std::chrono::high_resolution_clock::now();
	std::chrono::duration<double> time_analyser = t2 - t1;
	std::cout << "Cublas analyser on A used " << std::chrono::duration_cast<std::chrono::microseconds>(time_analyser).count() << " us." << std::endl;

	/* Create info objects for the ILU0 preconditioner */
	cusparseSolveAnalysisInfo_t info_u;
	cusparseCreateSolveAnalysisInfo(&info_u);

	hipsparseMatDescr_t descrL = 0;
	cusparseStatus = hipsparseCreateMatDescr(&descrL);
	hipsparseSetMatType(descrL, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descrL, HIPSPARSE_INDEX_BASE_ZERO);
	hipsparseSetMatFillMode(descrL, HIPSPARSE_FILL_MODE_LOWER);
	hipsparseSetMatDiagType(descrL, HIPSPARSE_DIAG_TYPE_UNIT);

	hipsparseMatDescr_t descrU = 0;
	cusparseStatus = hipsparseCreateMatDescr(&descrU);
	hipsparseSetMatType(descrU, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descrU, HIPSPARSE_INDEX_BASE_ZERO);
	hipsparseSetMatFillMode(descrU, HIPSPARSE_FILL_MODE_UPPER);
	hipsparseSetMatDiagType(descrU, HIPSPARSE_DIAG_TYPE_NON_UNIT);
	cusparseStatus = cusparseScsrsv_analysis(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, nz, descrU, d_val, d_row, d_col, info_u);

	/* reset the initial guess of the solution to zero */
	for (int i = 0; i < N; i++) x[i] = 0.0;

	hipMemcpy(d_r, rhs.data(), N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_x, x.data(), N * sizeof(float), hipMemcpyHostToDevice);

	t1 = std::chrono::high_resolution_clock::now();
	int k = 0;
	hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, &r1);

#ifdef CGTIMING
	double totalItTime, totalTriangularTime, totalSpmvTime;
	totalItTime = totalTriangularTime = totalSpmvTime = 0;
	hipEvent_t startTotal, stopTotal, startTri, stopTri, startSpmv, stopSpmv;
	hipEventCreate(&startTotal); hipEventCreate(&stopTotal);
	hipEventCreate(&startTri); hipEventCreate(&stopTri);
	hipEventCreate(&startSpmv); hipEventCreate(&stopSpmv);
#endif // CGTIMING

	while (r1 > tol*tol && k <= max_iter)
	{
#ifdef CGTIMING
		hipEventRecord(startTotal);
		hipEventRecord(startTri);
#endif // CGTIMING
		// Forward Solve, we can re-use infoA since the sparsity pattern of A matches that of L
		cusparseStatus = cusparseScsrsv_solve(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, &floatone, descrL,
			d_valsILU0, d_row, d_col, infoA, d_r, d_y);

		// Back Substitution
		cusparseStatus = cusparseScsrsv_solve(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, &floatone, descrU,
			d_valsILU0, d_row, d_col, info_u, d_y, d_zm1);
#ifdef CGTIMING
		hipEventRecord(stopTri);
#endif // CGTIMING

		k++;

		if (k == 1)
		{
			hipblasScopy(cublasHandle, N, d_zm1, 1, d_p, 1);
		}
		else
		{
			hipblasSdot(cublasHandle, N, d_r, 1, d_zm1, 1, &numerator);
			hipblasSdot(cublasHandle, N, d_rm2, 1, d_zm2, 1, &denominator);
			beta = numerator / denominator;
			hipblasSscal(cublasHandle, N, &beta, d_p, 1);
			hipblasSaxpy(cublasHandle, N, &floatone, d_zm1, 1, d_p, 1);
		}
#ifdef CGTIMING
		hipEventRecord(startSpmv);
#endif // CGTIMING
		hipsparseScsrmv(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, nzILU0, &floatone, descrU, d_val, d_row, d_col, d_p, &floatzero, d_omega);
#ifdef CGTIMING
		hipEventRecord(stopSpmv);
#endif // CGTIMING
		hipblasSdot(cublasHandle, N, d_r, 1, d_zm1, 1, &numerator);
		hipblasSdot(cublasHandle, N, d_p, 1, d_omega, 1, &denominator);
		alpha = numerator / denominator;
		hipblasSaxpy(cublasHandle, N, &alpha, d_p, 1, d_x, 1);
		hipblasScopy(cublasHandle, N, d_r, 1, d_rm2, 1);
		hipblasScopy(cublasHandle, N, d_zm1, 1, d_zm2, 1);
		nalpha = -alpha;
		hipblasSaxpy(cublasHandle, N, &nalpha, d_omega, 1, d_r, 1);
		hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, &r1);
#ifdef CGTIMING
		hipEventRecord(stopTotal);
#endif // CGTIMING
#ifdef CGTIMING
		hipEventSynchronize(stopTotal); hipEventSynchronize(stopTri); hipEventSynchronize(stopSpmv);
		float msTotal, msTri, msSpmv;  msTotal = msTri = msSpmv = 0;
		hipEventElapsedTime(&msTotal, startTotal, stopTotal); hipEventElapsedTime(&msTri, startTri, stopTri); hipEventElapsedTime(&msSpmv, startSpmv, stopSpmv);
		totalItTime += (float)(1e3 * msTotal);
		totalTriangularTime += (float)(1e3 * msTri);
		totalSpmvTime += (float)(1e3 *  msSpmv);
#endif // CGTIMING
	}
	hipMemcpy(x.data(), d_x, N * sizeof(float), hipMemcpyDeviceToHost); 
	t2 = std::chrono::high_resolution_clock::now();

	///************************/
	///* now write out result */
	///************************/
	std::chrono::duration<double> time_executor = t2 - t1;
	std::cout << "Cublas executor on A used " << std::chrono::duration_cast<std::chrono::microseconds>(time_executor).count() << " us. Final residual is " << sqrt(r1) << " after " << k << " iterations." << std::endl;

#ifdef CGTIMING
	totalItTime /= (double)k; totalTriangularTime /= (double)k; totalSpmvTime /= (double)k;
	std::cout << "Average cublas/cusparse iteration time breakdown: " << totalTriangularTime << " (triangular solver) " << totalSpmvTime << " (spmv) " << totalItTime - totalTriangularTime - totalSpmvTime << " (remaining) " << totalItTime << " (total)." << std::endl;
#endif // CGTIMING

	/* Destroy parameters */
	cusparseDestroySolveAnalysisInfo(infoA);
	cusparseDestroySolveAnalysisInfo(info_u);

	/* Destroy contexts */
	hipsparseDestroy(cusparseHandle);
	hipblasDestroy(cublasHandle);

	/* Free device memory */
	free(valsILU0);
	hipFree(d_col);
	hipFree(d_row);
	hipFree(d_val);
	hipFree(d_x);
	hipFree(d_y);
	hipFree(d_r);
	hipFree(d_p);
	hipFree(d_omega);
	hipFree(d_valsILU0);
	hipFree(d_zm1);
	hipFree(d_zm2);
	hipFree(d_rm2);

	return std::make_tuple(std::chrono::duration_cast<std::chrono::microseconds>(time_analyser).count(), std::chrono::duration_cast<std::chrono::microseconds>(time_executor).count());
}