#include "solvercublas.h"
#include <stdlib.h>
#include <stdio.h>
#include <fstream>

using namespace Cublas;

CG_Solver::~CG_Solver() {
	/* Destroy parameters */
	cusparseDestroySolveAnalysisInfo(infoA);
	cusparseDestroySolveAnalysisInfo(info_u);

	/* Destroy contexts */
	hipsparseDestroy(CusparseHandle::Instance().getHandle());
	hipblasDestroy(CublasHandle::Instance().getHandle());

	/* Free device memory */
	hipFree(d_col);
	hipFree(d_row);
	hipFree(d_val);
	hipFree(d_x);
	hipFree(d_y);
	hipFree(d_r);
	hipFree(d_p);
	hipFree(d_omega);
	hipFree(d_valsILU0);
	hipFree(d_zm1);
	hipFree(d_zm2);
	hipFree(d_rm2);
}

void CG_Solver::cudaInitialize(float *val, int M, int N, int nz, int *I, int *J, float *rhs) {
	this->M = M; this->N = N; this->nz = nz;
	x = (float *)malloc(sizeof(float)*N);
	for (int i = 0; i < N; i++) x[i] = 0.0;

	hipsparseStatus_t cusparseStatus;
	/* Description of the A matrix*/
	descr = 0;
	cusparseStatus = hipsparseCreateMatDescr(&descr);

	/* Define the properties of the matrix */
	hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

	/* Allocate required memory */
	hipMalloc((void **)&d_col, nz * sizeof(int));
	hipMalloc((void **)&d_row, (N + 1) * sizeof(int));
	hipMalloc((void **)&d_val, nz * sizeof(float));
	hipMalloc((void **)&d_x, N * sizeof(float));
	hipMalloc((void **)&d_y, N * sizeof(float));
	hipMalloc((void **)&d_r, N * sizeof(float));
	hipMalloc((void **)&d_p, N * sizeof(float));
	hipMalloc((void **)&d_omega, N * sizeof(float));

	hipMemcpy(d_col, J, nz * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_row, I, (N + 1) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_val, val, nz * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_r, rhs, N * sizeof(float), hipMemcpyHostToDevice);

	nzILU0 = 2 * N - 1;
	hipMalloc((void **)&d_valsILU0, nz * sizeof(float));
	hipMalloc((void **)&d_zm1, (N) * sizeof(float));
	hipMalloc((void **)&d_zm2, (N) * sizeof(float));
	hipMalloc((void **)&d_rm2, (N) * sizeof(float));
}

void CG_Solver::calculatePrecond() {
	hipsparseStatus_t cusparseStatus;
	/* create the analysis info object for the A matrix */
	infoA = 0;
	cusparseStatus = cusparseCreateSolveAnalysisInfo(&infoA);

	//checkCudaErrors(cusparseStatus);

	/* Perform the analysis for the Non-Transpose case */
	cusparseStatus = cusparseScsrsv_analysis(CusparseHandle::Instance().getHandle(), HIPSPARSE_OPERATION_NON_TRANSPOSE,
		N, nz, descr, d_val, d_row, d_col, infoA);

	//checkCudaErrors(cusparseStatus);

	/* Copy A data to ILU0 vals as input*/
	hipMemcpy(d_valsILU0, d_val, nz * sizeof(float), hipMemcpyDeviceToDevice);

	/* generate the Incomplete LU factor H for the matrix A using cudsparseScsrilu0 */
	cusparseStatus = cusparseScsrilu0(CusparseHandle::Instance().getHandle(), HIPSPARSE_OPERATION_NON_TRANSPOSE, N, descr, d_valsILU0, d_row, d_col, infoA);

	//checkCudaErrors(cusparseStatus);

	/* Create info objects for the ILU0 preconditioner */
	cusparseCreateSolveAnalysisInfo(&info_u);

	descrL = 0;
	cusparseStatus = hipsparseCreateMatDescr(&descrL);
	hipsparseSetMatType(descrL, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descrL, HIPSPARSE_INDEX_BASE_ZERO);
	hipsparseSetMatFillMode(descrL, HIPSPARSE_FILL_MODE_LOWER);
	hipsparseSetMatDiagType(descrL, HIPSPARSE_DIAG_TYPE_UNIT);

	descrU = 0;
	cusparseStatus = hipsparseCreateMatDescr(&descrU);
	hipsparseSetMatType(descrU, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descrU, HIPSPARSE_INDEX_BASE_ZERO);
	hipsparseSetMatFillMode(descrU, HIPSPARSE_FILL_MODE_UPPER);
	hipsparseSetMatDiagType(descrU, HIPSPARSE_DIAG_TYPE_NON_UNIT);
	cusparseStatus = cusparseScsrsv_analysis(CusparseHandle::Instance().getHandle(), HIPSPARSE_OPERATION_NON_TRANSPOSE, N, nz, descrU, d_val, d_row, d_col, info_u);

	k = 0;
	hipblasSdot(CublasHandle::Instance().getHandle(), N, d_r, 1, d_r, 1, &r1);
}

void CG_Solver::doIteration() {
	const float tol = 1e-12f;
	const int max_iter = 1000;
	const float floatone = 1.0;
	const float floatzero = 0.0;
	float alpha, beta;
	float numerator, denominator, nalpha;
	hipsparseStatus_t cusparseStatus;

	// Forward Solve, we can re-use infoA since the sparsity pattern of A matches that of L
	cusparseStatus = cusparseScsrsv_solve(CusparseHandle::Instance().getHandle(), HIPSPARSE_OPERATION_NON_TRANSPOSE, N, &floatone, descrL,
		d_valsILU0, d_row, d_col, infoA, d_r, d_y);
	//checkCudaErrors(cusparseStatus);

	// Back Substitution
	cusparseStatus = cusparseScsrsv_solve(CusparseHandle::Instance().getHandle(), HIPSPARSE_OPERATION_NON_TRANSPOSE, N, &floatone, descrU,
		d_valsILU0, d_row, d_col, info_u, d_y, d_zm1);
	//checkCudaErrors(cusparseStatus);

	k++;

	if (k == 1)
	{
		hipblasScopy(CublasHandle::Instance().getHandle(), N, d_zm1, 1, d_p, 1);
	}
	else
	{
		hipblasSdot(CublasHandle::Instance().getHandle(), N, d_r, 1, d_zm1, 1, &numerator);
		hipblasSdot(CublasHandle::Instance().getHandle(), N, d_rm2, 1, d_zm2, 1, &denominator);
		beta = numerator / denominator;
		hipblasSscal(CublasHandle::Instance().getHandle(), N, &beta, d_p, 1);
		hipblasSaxpy(CublasHandle::Instance().getHandle(), N, &floatone, d_zm1, 1, d_p, 1);
	}

	hipsparseScsrmv(CusparseHandle::Instance().getHandle(), HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, nzILU0, &floatone, descrU, d_val, d_row, d_col, d_p, &floatzero, d_omega);
	hipblasSdot(CublasHandle::Instance().getHandle(), N, d_r, 1, d_zm1, 1, &numerator);
	hipblasSdot(CublasHandle::Instance().getHandle(), N, d_p, 1, d_omega, 1, &denominator);
	alpha = numerator / denominator;
	hipblasSaxpy(CublasHandle::Instance().getHandle(), N, &alpha, d_p, 1, d_x, 1);
	hipblasScopy(CublasHandle::Instance().getHandle(), N, d_r, 1, d_rm2, 1);
	hipblasScopy(CublasHandle::Instance().getHandle(), N, d_zm1, 1, d_zm2, 1);
	nalpha = -alpha;
	hipblasSaxpy(CublasHandle::Instance().getHandle(), N, &nalpha, d_omega, 1, d_r, 1);
	hipblasSdot(CublasHandle::Instance().getHandle(), N, d_r, 1, d_r, 1, &r1);
}

float *CG_Solver::getX() {
	hipMemcpy(x, d_x, N * sizeof(float), hipMemcpyDeviceToHost);
	return x;
}


Precond *Precond::createPrecond(Eigen::SparseMatrix<float, 0, int> *A) {
	Precond *precond = new Precond;

	//hipsparseStatus_t cusparseStatus;
	///* create the analysis info object for the A matrix */
	//cusparseSolveAnalysisInfo_t infoA = 0;
	//cusparseStatus = cusparseCreateSolveAnalysisInfo(&infoA);

	////checkCudaErrors(cusparseStatus);

	///* Perform the analysis for the Non-Transpose case */
	//cusparseStatus = cusparseScsrsv_analysis(CusparseHandle::Instance().getHandle(), HIPSPARSE_OPERATION_NON_TRANSPOSE,
	//	N, nz, descr, d_val, d_row, d_col, infoA);

	////checkCudaErrors(cusparseStatus);

	///* Copy A data to ILU0 vals as input*/
	//hipMemcpy(d_valsILU0, d_val, nz * sizeof(float), hipMemcpyDeviceToDevice);

	///* generate the Incomplete LU factor H for the matrix A using cudsparseScsrilu0 */
	//cusparseStatus = cusparseScsrilu0(CusparseHandle::Instance().getHandle(), HIPSPARSE_OPERATION_NON_TRANSPOSE, N, descr, d_valsILU0, d_row, d_col, infoA);

	////checkCudaErrors(cusparseStatus);

	///* Create info objects for the ILU0 preconditioner */
	//cusparseCreateSolveAnalysisInfo(&info_u);

	//descrL = 0;
	//cusparseStatus = hipsparseCreateMatDescr(&descrL);
	//hipsparseSetMatType(descrL, HIPSPARSE_MATRIX_TYPE_GENERAL);
	//hipsparseSetMatIndexBase(descrL, HIPSPARSE_INDEX_BASE_ZERO);
	//hipsparseSetMatFillMode(descrL, HIPSPARSE_FILL_MODE_LOWER);
	//hipsparseSetMatDiagType(descrL, HIPSPARSE_DIAG_TYPE_UNIT);

	//descrU = 0;
	//cusparseStatus = hipsparseCreateMatDescr(&descrU);
	//hipsparseSetMatType(descrU, HIPSPARSE_MATRIX_TYPE_GENERAL);
	//hipsparseSetMatIndexBase(descrU, HIPSPARSE_INDEX_BASE_ZERO);
	//hipsparseSetMatFillMode(descrU, HIPSPARSE_FILL_MODE_UPPER);
	//hipsparseSetMatDiagType(descrU, HIPSPARSE_DIAG_TYPE_NON_UNIT);
	//cusparseStatus = cusparseScsrsv_analysis(CusparseHandle::Instance().getHandle(), HIPSPARSE_OPERATION_NON_TRANSPOSE, N, nz, descrU, d_val, d_row, d_col, info_u);

	return precond;
}

void Matrix::cudaMemcpyCublasMatrix(Matrix *A) {
	A->descr = 0;
	hipsparseCreateMatDescr(&A->descr);

	/* Define the properties of the matrix */
	hipsparseSetMatType(A->descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(A->descr, HIPSPARSE_INDEX_BASE_ZERO);

	/* Allocate required memory */
	int N = A->N, nz = A->nz;
	hipMalloc((void **)&A->d_row, (N + 1) * sizeof(int));
	hipMalloc((void **)&A->d_val, nz * sizeof(float));
	hipMemcpy(A->d_col, A->J, nz * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(A->d_row, A->I, (N + 1) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(A->d_val, A->val, nz * sizeof(float), hipMemcpyHostToDevice);

}

CusparseHandle::CusparseHandle() {
	hdl = 0;
	hipsparseStatus_t cusparseStatus = hipsparseCreate(&hdl);
}

CusparseHandle::~CusparseHandle() {
	hipsparseDestroy(hdl);
}

CublasHandle::CublasHandle() {
	hdl = 0;
	hipblasStatus_t hipblasStatus_t = hipblasCreate(&hdl);
}

CublasHandle::~CublasHandle() {
	hipblasDestroy(hdl);
}
