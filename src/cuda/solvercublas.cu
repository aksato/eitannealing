#include "solvercublas.h"
#include <stdlib.h>
#include <stdio.h>
#include <fstream>

using namespace Cublas;

CG_Solver::~CG_Solver() {
	/* Free device memory */
	hipFree(d_x);
	hipFree(d_y);
	hipFree(d_r);
	hipFree(d_p);
	hipFree(d_omega);
	hipFree(d_zm1);
	hipFree(d_zm2);
	hipFree(d_rm2);
}

CG_Solver::CG_Solver(Matrix *_A, double *_b, Precond *_precond) : A(_A), precond(_precond), buffer(NULL) {
	// Initialize x
	int N = A->N;
	x = (double *)malloc(sizeof(double)*N);
	for (int i = 0; i < N; i++) x[i] = 0.0;

	/* Allocate required memory */
	hipMalloc((void **)&d_x, N * sizeof(double));
	hipMalloc((void **)&d_r, N * sizeof(double));
	hipMemcpy(d_x, x, N * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_r, _b, N * sizeof(double), hipMemcpyHostToDevice);
	hipMalloc((void **)&d_y, N * sizeof(double));
	hipMalloc((void **)&d_p, N * sizeof(double));
	hipMalloc((void **)&d_omega, N * sizeof(double));
	hipMalloc((void **)&d_zm1, (N) * sizeof(double));
	hipMalloc((void **)&d_zm2, (N) * sizeof(double));
	hipMalloc((void **)&d_rm2, (N) * sizeof(double));

	/* Wrap raw data into cuSPARSE generic API objects */
	matA = NULL;
	hipsparseCreateCsr(&matA, N, N, A->nz, A->d_row, A->d_col, A->d_val,
									HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
									HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);
	vecp = NULL;
	hipsparseCreateDnVec(&vecp, N, d_p, HIP_R_64F);
	vecomega = NULL;
	hipsparseCreateDnVec(&vecomega, N, d_omega, HIP_R_64F);
	
	/* Allocate workspace for cuSPARSE */
	const double floatone = 1.0;
	const double floatzero = 0.0;
	size_t bufferSize = 0;
	size_t tmp = 0;
	int stmp = 0;
	hipsparseSpMV_bufferSize(
		CusparseHandle::Instance().getHandle(), HIPSPARSE_OPERATION_NON_TRANSPOSE, &floatone, matA, vecp,
		&floatzero, vecomega, HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT, &tmp);
	if (tmp > bufferSize) {
		bufferSize = stmp;
	}
	hipsparseDcsrilu02_bufferSize(
		CusparseHandle::Instance().getHandle(), N, A->nz, A->descr, A->d_val, A->d_row, A->d_col, precond->infoILU, &stmp);
	if (stmp > bufferSize) {
		bufferSize = stmp;
	}
	hipsparseDcsrsv2_bufferSize(
		CusparseHandle::Instance().getHandle(), HIPSPARSE_OPERATION_NON_TRANSPOSE, N, A->nz, precond->descrL, A->d_val,
		A->d_row, A->d_col, precond->infoL, &stmp);
	if (stmp > bufferSize) {
	bufferSize = stmp;
	}
	hipsparseDcsrsv2_bufferSize(
		CusparseHandle::Instance().getHandle(), HIPSPARSE_OPERATION_NON_TRANSPOSE, N, A->nz, precond->descrU, A->d_val,
		A->d_row, A->d_col, precond->infoU, &stmp);
	if (stmp > bufferSize) {
	bufferSize = stmp;
	}
	hipMalloc(&buffer, bufferSize);

	/* Perform analysis for ILU(0) */
	hipsparseDcsrilu02_analysis(
		CusparseHandle::Instance().getHandle(), A->N, A->nz, A->descr, A->d_val, A->d_row, A->d_col, precond->infoILU,
		HIPSPARSE_SOLVE_POLICY_USE_LEVEL, buffer);

	/* Copy A data to ILU(0) vals as input*/
	hipMemcpy(precond->d_valsILU0, A->d_val, A->nz * sizeof(double),
							hipMemcpyDeviceToDevice);

	/* generate the ILU(0) factors */
	hipsparseDcsrilu02(CusparseHandle::Instance().getHandle(), A->N, A->nz, A->descr, precond->d_valsILU0,
									A->d_row, A->d_col, precond->infoILU,
									HIPSPARSE_SOLVE_POLICY_USE_LEVEL, buffer);

	/* perform triangular solve analysis */
	hipsparseDcsrsv2_analysis(CusparseHandle::Instance().getHandle(), HIPSPARSE_OPERATION_NON_TRANSPOSE,
							 A->N, A->nz, precond->descrL, precond->d_valsILU0, A->d_row, A->d_col, precond->infoL,
							 HIPSPARSE_SOLVE_POLICY_USE_LEVEL, buffer);

	hipsparseDcsrsv2_analysis(CusparseHandle::Instance().getHandle(), HIPSPARSE_OPERATION_NON_TRANSPOSE,
							 A->N, A->nz, precond->descrU, precond->d_valsILU0, A->d_row, A->d_col, precond->infoU,
							 HIPSPARSE_SOLVE_POLICY_USE_LEVEL, buffer);

	k = 0;
	hipblasDdot(CublasHandle::Instance().getHandle(), N, d_r, 1, d_r, 1, &r1);
}

void CG_Solver::doIteration() {
	const double tol = 1e-12f;
	const int max_iter = 1000;
	const double floatone = 1.0;
	const double floatzero = 0.0;
	double alpha, beta;
	double numerator, denominator, nalpha;
	hipsparseStatus_t cusparseStatus;
	int N = A->N;

	// preconditioner application: d_zm1 = U^-1 L^-1 d_r
    cusparseStatus = hipsparseDcsrsv2_solve(
        CusparseHandle::Instance().getHandle(), HIPSPARSE_OPERATION_NON_TRANSPOSE, N, A->nz, &floatone,
        precond->descrL, precond->d_valsILU0, A->d_row, A->d_col, precond->infoL, d_r, d_y,
        HIPSPARSE_SOLVE_POLICY_USE_LEVEL, buffer);
	//checkCudaErrors(cusparseStatus);
	cusparseStatus = hipsparseDcsrsv2_solve(
        CusparseHandle::Instance().getHandle(), HIPSPARSE_OPERATION_NON_TRANSPOSE, N, A->nz, &floatone,
        precond->descrU, precond->d_valsILU0, A->d_row, A->d_col, precond->infoU, d_y, d_zm1,
        HIPSPARSE_SOLVE_POLICY_USE_LEVEL, buffer);
	//checkCudaErrors(cusparseStatus);

	k++;

	if (k == 1)
	{
		hipblasDcopy(CublasHandle::Instance().getHandle(), N, d_zm1, 1, d_p, 1);
	}
	else
	{
		hipblasDdot(CublasHandle::Instance().getHandle(), N, d_r, 1, d_zm1, 1, &numerator);
		hipblasDdot(CublasHandle::Instance().getHandle(), N, d_rm2, 1, d_zm2, 1, &denominator);
		beta = numerator / denominator;
		hipblasDscal(CublasHandle::Instance().getHandle(), N, &beta, d_p, 1);
		hipblasDaxpy(CublasHandle::Instance().getHandle(), N, &floatone, d_zm1, 1, d_p, 1);
	}

	hipsparseSpMV(
        CusparseHandle::Instance().getHandle(), HIPSPARSE_OPERATION_NON_TRANSPOSE, &floatone, matA, vecp,
        &floatzero, vecomega, HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT, buffer);
	hipblasDdot(CublasHandle::Instance().getHandle(), N, d_r, 1, d_zm1, 1, &numerator);
	hipblasDdot(CublasHandle::Instance().getHandle(), N, d_p, 1, d_omega, 1, &denominator);
	alpha = numerator / denominator;
	hipblasDaxpy(CublasHandle::Instance().getHandle(), N, &alpha, d_p, 1, d_x, 1);
	hipblasDcopy(CublasHandle::Instance().getHandle(), N, d_r, 1, d_rm2, 1);
	hipblasDcopy(CublasHandle::Instance().getHandle(), N, d_zm1, 1, d_zm2, 1);
	nalpha = -alpha;
	hipblasDaxpy(CublasHandle::Instance().getHandle(), N, &nalpha, d_omega, 1, d_r, 1);
	hipblasDdot(CublasHandle::Instance().getHandle(), N, d_r, 1, d_r, 1, &r1);
}

double *CG_Solver::getX() {
	hipMemcpy(x, d_x, A->N * sizeof(double), hipMemcpyDeviceToHost);
	return x;
}

Precond::~Precond() {
	/* Destroy parameters */
	hipsparseDestroyCsrilu02Info(infoILU);
	hipsparseDestroyMatDescr(descrL);
	hipsparseDestroyMatDescr(descrU);
}

Precond *Precond::createPrecond(Matrix *A) {
	Precond *precond = new Precond;
	hipsparseStatus_t cusparseStatus;
	
	/* Create ILU(0) info object */
	precond->infoILU = NULL;
	hipsparseCreateCsrilu02Info(&precond->infoILU);
	  
	/* Create L factor descriptor and triangular solve info */
	precond->descrL = NULL;
	cusparseStatus = hipsparseCreateMatDescr(&precond->descrL);
	hipsparseSetMatType(precond->descrL, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(precond->descrL, HIPSPARSE_INDEX_BASE_ZERO);
	hipsparseSetMatFillMode(precond->descrL, HIPSPARSE_FILL_MODE_LOWER);
	hipsparseSetMatDiagType(precond->descrL, HIPSPARSE_DIAG_TYPE_UNIT);
	precond->infoL = NULL;
	hipsparseCreateCsrsv2Info(&precond->infoL);

	precond->descrU = NULL;
	cusparseStatus = hipsparseCreateMatDescr(&precond->descrU);
	hipsparseSetMatType(precond->descrU, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(precond->descrU, HIPSPARSE_INDEX_BASE_ZERO);
	hipsparseSetMatFillMode(precond->descrU, HIPSPARSE_FILL_MODE_UPPER);
	hipsparseSetMatDiagType(precond->descrU, HIPSPARSE_DIAG_TYPE_NON_UNIT);
	precond->infoU = NULL;
	hipsparseCreateCsrsv2Info(&precond->infoU);

	return precond;
}

Matrix::~Matrix() {
	hipFree(d_col);
	hipFree(d_row);
	hipFree(d_val);
}

void Matrix::cudaMemcpyCublasMatrix(Matrix *A) {
	A->descr = 0;
	hipsparseCreateMatDescr(&A->descr);

	/* Define the properties of the matrix */
	hipsparseSetMatType(A->descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(A->descr, HIPSPARSE_INDEX_BASE_ZERO);

	/* Allocate required memory */
	int N = A->N, nz = A->nz;
	hipMalloc((void **)&A->d_col, nz * sizeof(int));
	hipMalloc((void **)&A->d_row, (N + 1) * sizeof(int));
	hipMalloc((void **)&A->d_val, nz * sizeof(double));
	hipMemcpy(A->d_col, A->J, nz * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(A->d_row, A->I, (N + 1) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(A->d_val, A->val, nz * sizeof(double), hipMemcpyHostToDevice);

}

CusparseHandle::CusparseHandle() {
	hdl = 0;
	hipsparseStatus_t cusparseStatus = hipsparseCreate(&hdl);
}

CusparseHandle::~CusparseHandle() {
	hipsparseDestroy(hdl);
}

CublasHandle::CublasHandle() {
	hdl = 0;
	hipblasStatus_t hipblasStatus_t = hipblasCreate(&hdl);
}

CublasHandle::~CublasHandle() {
	hipblasDestroy(hdl);
}
