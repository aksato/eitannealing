#include "solvercublas.h"
#include <stdlib.h>
#include <stdio.h>
#include <fstream>

using namespace Cublas;

CG_Solver::~CG_Solver() {
	/* Free device memory */
	hipFree(d_x);
	hipFree(d_y);
	hipFree(d_r);
	hipFree(d_p);
	hipFree(d_omega);
	hipFree(d_zm1);
	hipFree(d_zm2);
	hipFree(d_rm2);
}

CG_Solver::CG_Solver(Matrix *_A, float *_b, Precond *_precond) : A(_A), precond(_precond) {
	// Initialize x
	int N = A->N;
	x = (float *)malloc(sizeof(float)*N);
	for (int i = 0; i < N; i++) x[i] = 0.0;

	/* Allocate required memory */
	hipMalloc((void **)&d_x, N * sizeof(float));
	hipMalloc((void **)&d_r, N * sizeof(float));
	hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_r, _b, N * sizeof(float), hipMemcpyHostToDevice);
	hipMalloc((void **)&d_y, N * sizeof(float));
	hipMalloc((void **)&d_p, N * sizeof(float));
	hipMalloc((void **)&d_omega, N * sizeof(float));
	hipMalloc((void **)&d_zm1, (N) * sizeof(float));
	hipMalloc((void **)&d_zm2, (N) * sizeof(float));
	hipMalloc((void **)&d_rm2, (N) * sizeof(float));

	k = 0;
	hipblasSdot(CublasHandle::Instance().getHandle(), N, d_r, 1, d_r, 1, &r1);
}

void CG_Solver::doIteration() {
	const float tol = 1e-12f;
	const int max_iter = 1000;
	const float floatone = 1.0;
	const float floatzero = 0.0;
	float alpha, beta;
	float numerator, denominator, nalpha;
	hipsparseStatus_t cusparseStatus;
	int N = A->N;

	// Forward Solve, we can re-use infoA since the sparsity pattern of A matches that of L
	cusparseStatus = cusparseScsrsv_solve(CusparseHandle::Instance().getHandle(), HIPSPARSE_OPERATION_NON_TRANSPOSE, N, &floatone, precond->descrL,
		precond->d_valsILU0, A->d_row, A->d_col, precond->infoA, d_r, d_y);
	//checkCudaErrors(cusparseStatus);

	// Back Substitution
	cusparseStatus = cusparseScsrsv_solve(CusparseHandle::Instance().getHandle(), HIPSPARSE_OPERATION_NON_TRANSPOSE, N, &floatone, precond->descrU,
		precond->d_valsILU0, A->d_row, A->d_col, precond->info_u, d_y, d_zm1);
	//checkCudaErrors(cusparseStatus);

	k++;

	if (k == 1)
	{
		hipblasScopy(CublasHandle::Instance().getHandle(), N, d_zm1, 1, d_p, 1);
	}
	else
	{
		hipblasSdot(CublasHandle::Instance().getHandle(), N, d_r, 1, d_zm1, 1, &numerator);
		hipblasSdot(CublasHandle::Instance().getHandle(), N, d_rm2, 1, d_zm2, 1, &denominator);
		beta = numerator / denominator;
		hipblasSscal(CublasHandle::Instance().getHandle(), N, &beta, d_p, 1);
		hipblasSaxpy(CublasHandle::Instance().getHandle(), N, &floatone, d_zm1, 1, d_p, 1);
	}

	hipsparseScsrmv(CusparseHandle::Instance().getHandle(), HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, precond->nzILU0, &floatone, precond->descrU, A->d_val, A->d_row, A->d_col, d_p, &floatzero, d_omega);
	hipblasSdot(CublasHandle::Instance().getHandle(), N, d_r, 1, d_zm1, 1, &numerator);
	hipblasSdot(CublasHandle::Instance().getHandle(), N, d_p, 1, d_omega, 1, &denominator);
	alpha = numerator / denominator;
	hipblasSaxpy(CublasHandle::Instance().getHandle(), N, &alpha, d_p, 1, d_x, 1);
	hipblasScopy(CublasHandle::Instance().getHandle(), N, d_r, 1, d_rm2, 1);
	hipblasScopy(CublasHandle::Instance().getHandle(), N, d_zm1, 1, d_zm2, 1);
	nalpha = -alpha;
	hipblasSaxpy(CublasHandle::Instance().getHandle(), N, &nalpha, d_omega, 1, d_r, 1);
	hipblasSdot(CublasHandle::Instance().getHandle(), N, d_r, 1, d_r, 1, &r1);
}

float *CG_Solver::getX() {
	hipMemcpy(x, d_x, A->N * sizeof(float), hipMemcpyDeviceToHost);
	return x;
}

Precond::~Precond() {
	/* Destroy parameters */
	cusparseDestroySolveAnalysisInfo(infoA);
	cusparseDestroySolveAnalysisInfo(info_u);
	hipFree(d_valsILU0);
}

Precond *Precond::createPrecond(Matrix *A) {
	Precond *precond = new Precond;

	hipsparseStatus_t cusparseStatus;
	/* create the analysis info object for the A matrix */
	precond->infoA = 0;
	cusparseStatus = cusparseCreateSolveAnalysisInfo(&precond->infoA);

	/* Perform the analysis for the Non-Transpose case */
	cusparseStatus = cusparseScsrsv_analysis(CusparseHandle::Instance().getHandle(), HIPSPARSE_OPERATION_NON_TRANSPOSE,
		A->N, A->nz, A->descr, A->d_val, A->d_row, A->d_col, precond->infoA);

	/* Copy A data to ILU0 vals as input*/
	hipMalloc((void **)&precond->d_valsILU0, A->nz * sizeof(float));
	hipMemcpy(precond->d_valsILU0, A->d_val, A->nz * sizeof(float), hipMemcpyDeviceToDevice);

	/* generate the Incomplete LU factor H for the matrix A using cudsparseScsrilu0 */
	cusparseStatus = cusparseScsrilu0(CusparseHandle::Instance().getHandle(), HIPSPARSE_OPERATION_NON_TRANSPOSE, A->N, A->descr, precond->d_valsILU0, A->d_row, A->d_col, precond->infoA);

	///* Create info objects for the ILU0 preconditioner */
	cusparseCreateSolveAnalysisInfo(&precond->info_u);

	precond->descrL = 0;
	cusparseStatus = hipsparseCreateMatDescr(&precond->descrL);
	hipsparseSetMatType(precond->descrL, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(precond->descrL, HIPSPARSE_INDEX_BASE_ZERO);
	hipsparseSetMatFillMode(precond->descrL, HIPSPARSE_FILL_MODE_LOWER);
	hipsparseSetMatDiagType(precond->descrL, HIPSPARSE_DIAG_TYPE_UNIT);

	precond->descrU = 0;
	cusparseStatus = hipsparseCreateMatDescr(&precond->descrU);
	hipsparseSetMatType(precond->descrU, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(precond->descrU, HIPSPARSE_INDEX_BASE_ZERO);
	hipsparseSetMatFillMode(precond->descrU, HIPSPARSE_FILL_MODE_UPPER);
	hipsparseSetMatDiagType(precond->descrU, HIPSPARSE_DIAG_TYPE_NON_UNIT);
	cusparseStatus = cusparseScsrsv_analysis(CusparseHandle::Instance().getHandle(), HIPSPARSE_OPERATION_NON_TRANSPOSE, A->N, A->nz, precond->descrU, A->d_val, A->d_row, A->d_col, precond->info_u);

	precond->nzILU0 = 2 * A->N - 1;

	return precond;
}

Matrix::~Matrix() {
	hipFree(d_col);
	hipFree(d_row);
	hipFree(d_val);
}

void Matrix::cudaMemcpyCublasMatrix(Matrix *A) {
	A->descr = 0;
	hipsparseCreateMatDescr(&A->descr);

	/* Define the properties of the matrix */
	hipsparseSetMatType(A->descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(A->descr, HIPSPARSE_INDEX_BASE_ZERO);

	/* Allocate required memory */
	int N = A->N, nz = A->nz;
	hipMalloc((void **)&A->d_col, nz * sizeof(int));
	hipMalloc((void **)&A->d_row, (N + 1) * sizeof(int));
	hipMalloc((void **)&A->d_val, nz * sizeof(float));
	hipMemcpy(A->d_col, A->J, nz * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(A->d_row, A->I, (N + 1) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(A->d_val, A->val, nz * sizeof(float), hipMemcpyHostToDevice);

}

CusparseHandle::CusparseHandle() {
	hdl = 0;
	hipsparseStatus_t cusparseStatus = hipsparseCreate(&hdl);
}

CusparseHandle::~CusparseHandle() {
	hipsparseDestroy(hdl);
}

CublasHandle::CublasHandle() {
	hdl = 0;
	hipblasStatus_t hipblasStatus_t = hipblasCreate(&hdl);
}

CublasHandle::~CublasHandle() {
	hipblasDestroy(hdl);
}
