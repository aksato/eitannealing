#include "solvercublas.h"
#include <stdlib.h>
#include <stdio.h>
#include <fstream>

CGCUBLAS_Solver::~CGCUBLAS_Solver() {
	/* Destroy parameters */
	cusparseDestroySolveAnalysisInfo(infoA);
	cusparseDestroySolveAnalysisInfo(info_u);

	/* Destroy contexts */
	hipsparseDestroy(cusparseHandle);
	hipblasDestroy(cublasHandle);

	/* Free device memory */
	hipFree(d_col);
	hipFree(d_row);
	hipFree(d_val);
	hipFree(d_x);
	hipFree(d_y);
	hipFree(d_r);
	hipFree(d_p);
	hipFree(d_omega);
	hipFree(d_valsILU0);
	hipFree(d_zm1);
	hipFree(d_zm2);
	hipFree(d_rm2);
}

void CGCUBLAS_Solver::cudaInitialize(float *val, int M, int N, int nz, int *I, int *J, float *rhs) {
	this->M = M; this->N = N; this->nz = nz;
	x = (float *)malloc(sizeof(float)*N);
	for (int i = 0; i < N; i++) x[i] = 0.0;

	/* Create CUBLAS context */
	cublasHandle = 0;
	hipblasStatus_t hipblasStatus_t;
	hipblasStatus_t = hipblasCreate(&cublasHandle);

	/* Create CUSPARSE context */
	cusparseHandle = 0;
	hipsparseStatus_t cusparseStatus;
	cusparseStatus = hipsparseCreate(&cusparseHandle);

	/* Description of the A matrix*/
	descr = 0;
	cusparseStatus = hipsparseCreateMatDescr(&descr);

	/* Define the properties of the matrix */
	hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

	/* Allocate required memory */
	hipMalloc((void **)&d_col, nz * sizeof(int));
	hipMalloc((void **)&d_row, (N + 1) * sizeof(int));
	hipMalloc((void **)&d_val, nz * sizeof(float));
	hipMalloc((void **)&d_x, N * sizeof(float));
	hipMalloc((void **)&d_y, N * sizeof(float));
	hipMalloc((void **)&d_r, N * sizeof(float));
	hipMalloc((void **)&d_p, N * sizeof(float));
	hipMalloc((void **)&d_omega, N * sizeof(float));

	hipMemcpy(d_col, J, nz * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_row, I, (N + 1) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_val, val, nz * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_r, rhs, N * sizeof(float), hipMemcpyHostToDevice);

	nzILU0 = 2 * N - 1;
	hipMalloc((void **)&d_valsILU0, nz * sizeof(float));
	hipMalloc((void **)&d_zm1, (N) * sizeof(float));
	hipMalloc((void **)&d_zm2, (N) * sizeof(float));
	hipMalloc((void **)&d_rm2, (N) * sizeof(float));
}

void CGCUBLAS_Solver::calculatePrecond() {
	hipsparseStatus_t cusparseStatus;
	/* create the analysis info object for the A matrix */
	infoA = 0;
	cusparseStatus = cusparseCreateSolveAnalysisInfo(&infoA);

	//checkCudaErrors(cusparseStatus);

	/* Perform the analysis for the Non-Transpose case */
	cusparseStatus = cusparseScsrsv_analysis(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
		N, nz, descr, d_val, d_row, d_col, infoA);

	//checkCudaErrors(cusparseStatus);

	/* Copy A data to ILU0 vals as input*/
	hipMemcpy(d_valsILU0, d_val, nz * sizeof(float), hipMemcpyDeviceToDevice);

	/* generate the Incomplete LU factor H for the matrix A using cudsparseScsrilu0 */
	cusparseStatus = cusparseScsrilu0(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, descr, d_valsILU0, d_row, d_col, infoA);

	//checkCudaErrors(cusparseStatus);

	/* Create info objects for the ILU0 preconditioner */
	cusparseCreateSolveAnalysisInfo(&info_u);

	descrL = 0;
	cusparseStatus = hipsparseCreateMatDescr(&descrL);
	hipsparseSetMatType(descrL, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descrL, HIPSPARSE_INDEX_BASE_ZERO);
	hipsparseSetMatFillMode(descrL, HIPSPARSE_FILL_MODE_LOWER);
	hipsparseSetMatDiagType(descrL, HIPSPARSE_DIAG_TYPE_UNIT);

	descrU = 0;
	cusparseStatus = hipsparseCreateMatDescr(&descrU);
	hipsparseSetMatType(descrU, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descrU, HIPSPARSE_INDEX_BASE_ZERO);
	hipsparseSetMatFillMode(descrU, HIPSPARSE_FILL_MODE_UPPER);
	hipsparseSetMatDiagType(descrU, HIPSPARSE_DIAG_TYPE_NON_UNIT);
	cusparseStatus = cusparseScsrsv_analysis(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, nz, descrU, d_val, d_row, d_col, info_u);

	k = 0;
	hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, &r1);
}

void CGCUBLAS_Solver::doIteration() {
	const float tol = 1e-12f;
	const int max_iter = 1000;
	const float floatone = 1.0;
	const float floatzero = 0.0;
	float alpha, beta;
	float numerator, denominator, nalpha;
	hipsparseStatus_t cusparseStatus;

	// Forward Solve, we can re-use infoA since the sparsity pattern of A matches that of L
	cusparseStatus = cusparseScsrsv_solve(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, &floatone, descrL,
		d_valsILU0, d_row, d_col, infoA, d_r, d_y);
	//checkCudaErrors(cusparseStatus);

	// Back Substitution
	cusparseStatus = cusparseScsrsv_solve(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, &floatone, descrU,
		d_valsILU0, d_row, d_col, info_u, d_y, d_zm1);
	//checkCudaErrors(cusparseStatus);

	k++;

	if (k == 1)
	{
		hipblasScopy(cublasHandle, N, d_zm1, 1, d_p, 1);
	}
	else
	{
		hipblasSdot(cublasHandle, N, d_r, 1, d_zm1, 1, &numerator);
		hipblasSdot(cublasHandle, N, d_rm2, 1, d_zm2, 1, &denominator);
		beta = numerator / denominator;
		hipblasSscal(cublasHandle, N, &beta, d_p, 1);
		hipblasSaxpy(cublasHandle, N, &floatone, d_zm1, 1, d_p, 1);
	}

	hipsparseScsrmv(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, nzILU0, &floatone, descrU, d_val, d_row, d_col, d_p, &floatzero, d_omega);
	hipblasSdot(cublasHandle, N, d_r, 1, d_zm1, 1, &numerator);
	hipblasSdot(cublasHandle, N, d_p, 1, d_omega, 1, &denominator);
	alpha = numerator / denominator;
	hipblasSaxpy(cublasHandle, N, &alpha, d_p, 1, d_x, 1);
	hipblasScopy(cublasHandle, N, d_r, 1, d_rm2, 1);
	hipblasScopy(cublasHandle, N, d_zm1, 1, d_zm2, 1);
	nalpha = -alpha;
	hipblasSaxpy(cublasHandle, N, &nalpha, d_omega, 1, d_r, 1);
	hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, &r1);
}

float *CGCUBLAS_Solver::getX() {
	hipMemcpy(x, d_x, N * sizeof(float), hipMemcpyDeviceToHost);
	return x;
}


CGCUBLAS_Precond *CGCUBLAS_Precond::createPrecond(Eigen::SparseMatrix<float, 0, int> *A) {
	CGCUBLAS_Precond *precond = new CGCUBLAS_Precond;

	//hipsparseStatus_t cusparseStatus;
	///* create the analysis info object for the A matrix */
	//cusparseSolveAnalysisInfo_t infoA = 0;
	//cusparseStatus = cusparseCreateSolveAnalysisInfo(&infoA);

	////checkCudaErrors(cusparseStatus);

	///* Perform the analysis for the Non-Transpose case */
	//cusparseStatus = cusparseScsrsv_analysis(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
	//	N, nz, descr, d_val, d_row, d_col, infoA);

	////checkCudaErrors(cusparseStatus);

	///* Copy A data to ILU0 vals as input*/
	//hipMemcpy(d_valsILU0, d_val, nz * sizeof(float), hipMemcpyDeviceToDevice);

	///* generate the Incomplete LU factor H for the matrix A using cudsparseScsrilu0 */
	//cusparseStatus = cusparseScsrilu0(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, descr, d_valsILU0, d_row, d_col, infoA);

	////checkCudaErrors(cusparseStatus);

	///* Create info objects for the ILU0 preconditioner */
	//cusparseCreateSolveAnalysisInfo(&info_u);

	//descrL = 0;
	//cusparseStatus = hipsparseCreateMatDescr(&descrL);
	//hipsparseSetMatType(descrL, HIPSPARSE_MATRIX_TYPE_GENERAL);
	//hipsparseSetMatIndexBase(descrL, HIPSPARSE_INDEX_BASE_ZERO);
	//hipsparseSetMatFillMode(descrL, HIPSPARSE_FILL_MODE_LOWER);
	//hipsparseSetMatDiagType(descrL, HIPSPARSE_DIAG_TYPE_UNIT);

	//descrU = 0;
	//cusparseStatus = hipsparseCreateMatDescr(&descrU);
	//hipsparseSetMatType(descrU, HIPSPARSE_MATRIX_TYPE_GENERAL);
	//hipsparseSetMatIndexBase(descrU, HIPSPARSE_INDEX_BASE_ZERO);
	//hipsparseSetMatFillMode(descrU, HIPSPARSE_FILL_MODE_UPPER);
	//hipsparseSetMatDiagType(descrU, HIPSPARSE_DIAG_TYPE_NON_UNIT);
	//cusparseStatus = cusparseScsrsv_analysis(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, nz, descrU, d_val, d_row, d_col, info_u);

	return precond;
}

void CGCUBLAS_Matrix::cudaMemcpyCublasMatrix(CGCUBLAS_Matrix *A) {
	A->descr = 0;
	hipsparseCreateMatDescr(&A->descr);

	/* Define the properties of the matrix */
	hipsparseSetMatType(A->descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(A->descr, HIPSPARSE_INDEX_BASE_ZERO);

	/* Allocate required memory */
	int N = A->N, nz = A->nz;
	hipMalloc((void **)&A->d_row, (N + 1) * sizeof(int));
	hipMalloc((void **)&A->d_val, nz * sizeof(float));
	hipMemcpy(A->d_col, A->J, nz * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(A->d_row, A->I, (N + 1) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(A->d_val, A->val, nz * sizeof(float), hipMemcpyHostToDevice);

}