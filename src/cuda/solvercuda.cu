#include "solvercuda.h"
#include "vector.h"
#include "matrix-cpjds.h"
#include "solver-pcg.h"
#include "../nodecoefficients.h"

using namespace cgl;

CGCUDA_Solver::CGCUDA_Solver(numType *A, numType *b, nodeCoefficients **nodeCoef, int nodesCount, int numcoefficients, numType *precond, int n) {
	MatrixCPJDS *stiffness = new MatrixCPJDS;
	mgr = new MatrixCPJDSManager(A, n);
	mgr->buidMatrixCPJDS(stiffness, nodeCoef, nodesCount, numcoefficients);
	size = stiffness->matrixData.n;
	m_preconditioner_eigen(*stiffness, stiffness->cpuData.data, stiffness->cpuData.precond); // FIXME: Use already implemented preconditioner
	hipMemcpy(stiffness->preconditionedData, stiffness->cpuData.precond, (size_t)stiffness->matrixData.elCount * sizeof(numType), hipMemcpyHostToDevice);
	Vector *bVec = createCurrentVector(b, *mgr, size, n);
	solver = new PCGSolverCPJDS(mgr, stiffness, bVec);
	solver->init();
}

CGCUDA_Solver::CGCUDA_Solver(MatrixCPJDS *stiffness, MatrixCPJDSManager *mgr, Vector *bVec) : mgr(mgr) {
	size = stiffness->matrixData.n;
	solver = new PCGSolverCPJDS(mgr, stiffness, bVec);
	solver->init();
}

Vector *CGCUDA_Solver::createCurrentVector(numType *vec, MatrixCPJDSManager &mgr, int size, int n) {
	numType * vecArr = new numType[size];
	for (int i = 0; i < size; i++) {
		vecArr[i] = 0;
	}
	for (int i = 0; i < n; i++) {
		vecArr[mgr.original2PaddedIdx[i]] = vec[i];
	}
	return new Vector(vecArr, size);
}

void CGCUDA_Solver::doIteration() {
	solver->doIteration();
	hipDeviceSynchronize();
}

std::vector<numType> CGCUDA_Solver::getX() {
	Vector *x = solver->getX();
	
	return mgr->restore(x);
}

MatrixCPJDSManager *CGCUDA_Solver::createManager(numType * A, MatrixCPJDS *stiffness, nodeCoefficients **nodeCoef, int nodesCount, int numcoefficients, int n) {
	MatrixCPJDSManager *mgr = new MatrixCPJDSManager(A, n);
	mgr->buidMatrixCPJDS(stiffness, nodeCoef, nodesCount, numcoefficients);
	m_preconditioner_eigen(*stiffness, stiffness->cpuData.data, stiffness->cpuData.precond); // FIXME: Use already implemented preconditioner
	hipMemcpy(stiffness->preconditionedData, stiffness->cpuData.precond, (size_t)stiffness->matrixData.elCount * sizeof(numType), hipMemcpyHostToDevice);
	return mgr;
}

MatrixCPJDSManager *CGCUDA_Solver::createManager(Eigen::SparseMatrix<double> *A, MatrixCPJDS *stiffness, nodeCoefficients **nodeCoef, int nodesCount, int numcoefficients) {
	MatrixCPJDSManager *mgr = new MatrixCPJDSManager(A);
	mgr->buidMatrixCPJDS(stiffness, nodeCoef, nodesCount, numcoefficients);
	m_preconditioner_eigen(*stiffness, stiffness->cpuData.data, stiffness->cpuData.precond); // FIXME: Use already implemented preconditioner
	hipMemcpy(stiffness->preconditionedData, stiffness->cpuData.precond, (size_t)stiffness->matrixData.elCount * sizeof(numType), hipMemcpyHostToDevice);
	return mgr;
}