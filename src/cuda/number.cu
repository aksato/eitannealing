#include "number.h"

#ifdef USECUDA

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

/*****************************************************************************************/
/*****************************************************************************************/
/**********************************    functions    **************************************/
/*****************************************************************************************/
/*****************************************************************************************/

Number::Number(numType val) {
	// CUDA device memory allocation
	hipMalloc((void**)& this->data, sizeof (numType));
	// CUDA memory copy
	hipMemcpy(this->data, &val, (size_t) sizeof (numType), hipMemcpyHostToDevice);
}
Number::~Number() {
	hipFree(data);
}
void Number::copy(Number * src) {
	// CUDA memory copy
	hipMemcpy(this->data, src->data, (size_t) sizeof (numType), hipMemcpyHostToDevice);
}
void Number::copy(Number * src, hipStream_t stream) {
	// CUDA memory copy
	hipMemcpyAsync(this->data, src->data, (size_t) sizeof (numType), hipMemcpyHostToDevice, stream);
}

numType Number::transf2CPU() {
	numType *val = new numType[1];
	hipMemcpy(val, data, (size_t)sizeof(numType), hipMemcpyDeviceToHost);
	return val[0];
}

#endif