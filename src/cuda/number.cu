#include "number.h"

#ifdef USECUDA

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

/*****************************************************************************************/
/*****************************************************************************************/
/**********************************    functions    **************************************/
/*****************************************************************************************/
/*****************************************************************************************/

Number::Number(double val) {
	// CUDA device memory allocation
	hipMalloc((void**)& this->data, (size_t) sizeof (double));
	// CUDA memory copy
	hipMemcpy(this->data, &val, (size_t) sizeof (double), hipMemcpyHostToDevice);
}
Number::~Number() {
	hipFree(this->data);
}
void Number::copy(Number * src) {
	// CUDA memory copy
	hipMemcpy(this->data, src->data, (size_t) sizeof (double), hipMemcpyHostToDevice);
}
void Number::copy(Number * src, hipStream_t stream) {
	// CUDA memory copy
	hipMemcpyAsync(this->data, src->data, (size_t) sizeof (double), hipMemcpyHostToDevice, stream);
}

double Number::transf2CPU() {
	double val;
	hipMemcpy(&val, this->data, (size_t)sizeof(double), hipMemcpyDeviceToHost);
	return val;
}

#endif