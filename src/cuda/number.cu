#include "number.h"

#ifdef USECUDA

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

/*****************************************************************************************/
/*****************************************************************************************/
/**********************************    functions    **************************************/
/*****************************************************************************************/
/*****************************************************************************************/

Number::Number(double val) {
	// CUDA device memory allocation
	hipMalloc((void**)& this->data, sizeof (double));
	// CUDA memory copy
	hipMemcpy(this->data, &val, (size_t) sizeof (double), hipMemcpyHostToDevice);
}
Number::~Number() {
	hipFree(data);
}
void Number::copy(Number * src) {
	// CUDA memory copy
	hipMemcpy(this->data, src->data, (size_t) sizeof (double), hipMemcpyHostToDevice);
}
void Number::copy(Number * src, hipStream_t stream) {
	// CUDA memory copy
	hipMemcpyAsync(this->data, src->data, (size_t) sizeof (double), hipMemcpyHostToDevice, stream);
}

double Number::transf2CPU() {
	double *val = new double[1];
	hipMemcpy(val, data, (size_t)sizeof(double), hipMemcpyDeviceToHost);
	return val[0];
}

#endif